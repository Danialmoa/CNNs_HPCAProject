#include "hip/hip_runtime.h"
#include "../include/conv_block.cuh"
#include <iostream>
#include <vector>
#include <iomanip>
#include <numeric>

// Add function declarations at the top of the file
float calculate_mean(const float* data, int size);
float calculate_std(const float* data, int size);

// Helper function to print 3D tensor (channel, height, width)
void print_tensor(const float* data, int channels, int height, int width, const std::string& name) {
    std::cout << "\n" << name << " [" << channels << ", " << height << ", " << width << "]:\n";
    for (int c = 0; c < channels; c++) {
        std::cout << "Channel " << c << ":\n";
        for (int h = 0; h < height; h++) {
            for (int w = 0; w < width; w++) {
                int idx = (c * height + h) * width + w;
                std::cout << std::setw(6) << std::fixed << std::setprecision(2) 
                         << data[idx] << " ";
            }
            std::cout << "\n";
        }
        std::cout << "\n";
    }
}

// Simple MSE loss calculation
float calculate_loss(const float* output, const float* target, int size) {
    float loss = 0.0f;
    for (int i = 0; i < size; i++) {
        float diff = output[i] - target[i];
        loss += diff * diff;
    }
    return loss / size;
}

// Modify the input data initialization
float* generate_test_input(int batch_size, int channels, int height, int width) {
    float* data = new float[batch_size * channels * height * width];
    for (int b = 0; b < batch_size; b++) {
        for (int c = 0; c < channels; c++) {
            for (int h = 0; h < height; h++) {
                for (int w = 0; w < width; w++) {
                    int idx = ((b * channels + c) * height + h) * width + w;
                    // Generate different patterns for each channel
                    switch(c) {
                        case 0:
                            data[idx] = (h + w) % 2 == 0 ? 1.0f : 0.0f; // Checkerboard
                            break;
                        case 1:
                            data[idx] = float(h) / height; // Vertical gradient
                            break;
                        case 2:
                            data[idx] = float(w) / width; // Horizontal gradient
                            break;
                    }
                }
            }
        }
    }
    return data;
}

int main() {
    try {
        // Test parameters - increased batch size to better demonstrate stream benefits
        const int batch_size = 32;  // Increased from 2 to better show stream benefits
        const int in_channels = 3;
        const int hidden_channels = 8;
        const int out_channels = 4;
        const int height = 32;
        const int width = 32;
        const int kernel_size = 3;
        const int stride = 1;
        const int padding = 1;
        const int pool_size = 2;
        const int pool_stride = 2;
        const float learning_rate = 0.01f;
        const int num_epochs = 2;

        // Calculate output dimensions for conv1
        const int conv1_out_height = (height + 2 * padding - kernel_size) / stride + 1;
        const int conv1_out_width = (width + 2 * padding - kernel_size) / stride + 1;
        const int pool1_out_height = (conv1_out_height - pool_size) / pool_stride + 1;
        const int pool1_out_width = (conv1_out_width - pool_size) / pool_stride + 1;

        // Calculate output dimensions for conv2
        const int conv2_out_height = (pool1_out_height + 2 * padding - kernel_size) / stride + 1;
        const int conv2_out_width = (pool1_out_width + 2 * padding - kernel_size) / stride + 1;
        const int pool2_out_height = (conv2_out_height - pool_size) / pool_stride + 1;
        const int pool2_out_width = (conv2_out_width - pool_size) / pool_stride + 1;

        // Create ConvBlocks
        ConvBlock conv1(in_channels, hidden_channels, kernel_size, 
                       stride, padding, pool_size, pool_stride, 
                       learning_rate);
        
        ConvBlock conv2(hidden_channels, out_channels, kernel_size, 
                       stride, padding, pool_size, pool_stride, 
                       learning_rate);

        // Print architecture info
        std::cout << "\nNetwork Architecture:";
        std::cout << "\nInput: " << batch_size << "x" << in_channels << "x" << height << "x" << width;
        std::cout << "\nConv1: " << hidden_channels << " channels, " << pool1_out_height << "x" << pool1_out_width;
        std::cout << "\nConv2: " << out_channels << " channels, " << pool2_out_height << "x" << pool2_out_width;
        std::cout << "\n";

        // Add timing measurements
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Add more detailed logging
        std::cout << "\nInitial parameters:";
        std::cout << "\nConv1 Weights (mean, std): " 
                  << calculate_mean(conv1.get_weights(), hidden_channels * in_channels * kernel_size * kernel_size) << ", "
                  << calculate_std(conv1.get_weights(), hidden_channels * in_channels * kernel_size * kernel_size);
        std::cout << "\nConv2 Weights (mean, std): " 
                  << calculate_mean(conv2.get_weights(), out_channels * hidden_channels * kernel_size * kernel_size) << ", "
                  << calculate_std(conv2.get_weights(), out_channels * hidden_channels * kernel_size * kernel_size) << std::endl;

        // Training loop
        for (int epoch = 0; epoch < num_epochs; ++epoch) {
            std::cout << "\n=== Epoch " << epoch + 1 << " ===\n";

            // Create input data
            float* h_input = generate_test_input(batch_size, in_channels, height, width);

            // Create target data
            std::vector<float> h_target(batch_size * out_channels * pool2_out_height * pool2_out_width, 1.0f);

            // Allocate device memory with proper stream synchronization
            float *d_input, *d_conv1_output, *d_final_output;
            hipMalloc(&d_input, batch_size * in_channels * height * width * sizeof(float));
            hipMalloc(&d_conv1_output, batch_size * hidden_channels * pool1_out_height * pool1_out_width * sizeof(float));
            hipMalloc(&d_final_output, batch_size * out_channels * pool2_out_height * pool2_out_width * sizeof(float));

            // Copy input to device
            hipMemcpy(d_input, h_input, batch_size * in_channels * height * width * sizeof(float), hipMemcpyHostToDevice);

            // Time the forward pass
            hipEventRecord(start);
            
            // Forward pass
            conv1.forward(d_input, d_conv1_output, batch_size, height, width);
            conv2.forward(d_conv1_output, d_final_output, batch_size, pool1_out_height, pool1_out_width);
            
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            std::cout << "Forward pass time: " << milliseconds << " ms\n";

            // Get output and calculate loss
            std::vector<float> h_output(batch_size * out_channels * pool2_out_height * pool2_out_width);
            hipMemcpy(h_output.data(), d_final_output, h_output.size() * sizeof(float), hipMemcpyDeviceToHost);

            float loss = calculate_loss(h_output.data(), h_target.data(), h_output.size());
            std::cout << "Loss: " << loss << std::endl;

            // Print only first batch results to avoid overwhelming output
            if (epoch == 0) {
                print_tensor(h_input, in_channels, height, width, "Input (first batch)");
                print_tensor(h_output.data(), out_channels, pool2_out_height, pool2_out_width, 
                            "Output (first batch)");
            }

            // Time the backward pass
            hipEventRecord(start);

            // Backward pass with proper stream synchronization
            float *d_grad_output, *d_grad_conv1_output, *d_grad_input;
            hipMalloc(&d_grad_output, h_output.size() * sizeof(float));
            hipMalloc(&d_grad_conv1_output, batch_size * hidden_channels * pool1_out_height * pool1_out_width * sizeof(float));
            hipMalloc(&d_grad_input, batch_size * in_channels * height * width * sizeof(float));

            // Compute gradients
            std::vector<float> h_grad_output(h_output.size());
            for (size_t i = 0; i < h_output.size(); i++) {
                h_grad_output[i] = h_output[i] - h_target[i];
            }
            hipMemcpy(d_grad_output, h_grad_output.data(), h_grad_output.size() * sizeof(float), 
                      hipMemcpyHostToDevice);

            conv2.backward(d_grad_output, d_grad_conv1_output, batch_size, pool1_out_height, pool1_out_width);
            conv1.backward(d_grad_conv1_output, d_grad_input, batch_size, height, width);

            hipEventRecord(stop);
            hipEventSynchronize(stop);
            
            milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            std::cout << "Backward pass time: " << milliseconds << " ms\n";

            // Get gradients and updated parameters
            std::vector<float> h_grad_input(batch_size * in_channels * height * width);
            hipMemcpy(h_grad_input.data(), d_grad_input, h_grad_input.size() * sizeof(float), 
                      hipMemcpyDeviceToHost);

            // Print gradients
            print_tensor(h_grad_input.data(), in_channels, height, width, 
                        "Gradient w.r.t Input (first batch)");

            // Print updated parameters
            std::cout << "\nUpdated Parameters after epoch " << epoch + 1 << ":";
            
            // Conv1 parameters
            std::vector<float> h_conv1_weights(hidden_channels * in_channels * kernel_size * kernel_size);
            std::vector<float> h_conv1_biases(hidden_channels);
            hipMemcpy(h_conv1_weights.data(), conv1.get_weights(), 
                      h_conv1_weights.size() * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(h_conv1_biases.data(), conv1.get_biases(), 
                      h_conv1_biases.size() * sizeof(float), hipMemcpyDeviceToHost);

            // Conv2 parameters
            std::vector<float> h_conv2_weights(out_channels * hidden_channels * kernel_size * kernel_size);
            std::vector<float> h_conv2_biases(out_channels);
            hipMemcpy(h_conv2_weights.data(), conv2.get_weights(), 
                      h_conv2_weights.size() * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(h_conv2_biases.data(), conv2.get_biases(), 
                      h_conv2_biases.size() * sizeof(float), hipMemcpyDeviceToHost);

            // Print parameter statistics
            std::cout << "\nConv1 Weights (mean): " 
                     << std::accumulate(h_conv1_weights.begin(), h_conv1_weights.end(), 0.0f) / h_conv1_weights.size();
            std::cout << "\nConv2 Weights (mean): " 
                     << std::accumulate(h_conv2_weights.begin(), h_conv2_weights.end(), 0.0f) / h_conv2_weights.size();

            // Cleanup with proper synchronization
            hipDeviceSynchronize();
            hipFree(d_input);
            hipFree(d_conv1_output);
            hipFree(d_final_output);
            hipFree(d_grad_output);
            hipFree(d_grad_conv1_output);
            hipFree(d_grad_input);

            // Error checking
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                throw std::runtime_error(std::string("CUDA error: ") + hipGetErrorString(err));
            }

            std::cout << "\nEpoch " << epoch + 1 << " completed successfully!\n";
        }

        // Cleanup timing events
        hipEventDestroy(start);
        hipEventDestroy(stop);

        std::cout << "\nAll epochs completed successfully!\n";
        
    } catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
        return 1;
    }

    return 0;
}

// Function implementations remain at the bottom
float calculate_mean(const float* data, int size) {
    float sum = 0.0f;
    for (int i = 0; i < size; i++) {
        sum += data[i];
    }
    return sum / size;
}

float calculate_std(const float* data, int size) {
    float mean = calculate_mean(data, size);
    float sum_sq = 0.0f;
    for (int i = 0; i < size; i++) {
        float diff = data[i] - mean;
        sum_sq += diff * diff;
    }
    return std::sqrt(sum_sq / size);
}
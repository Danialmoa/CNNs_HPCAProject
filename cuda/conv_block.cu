#include "hip/hip_runtime.h"
#include "include/conv_block.cuh"
#include <random>
#include <cmath>
#include <stdexcept>
#include <iostream>

#define DEBUG_PRINT 1

// CUDA kernels
__global__ void conv_forward_kernel(
    const float* input, const float* weights, const float* biases,
    float* conv_output, float* relu_output,
    int batch_size, int in_channels, int out_channels,
    int height, int width, int kernel_size, int stride, int padding,
    int output_height, int output_width) {
    
    int b = blockIdx.x; // Batch Index
    int oc = blockIdx.y; // output channels
    int h = blockIdx.z / output_width; // height
    int w = blockIdx.z % output_width; // width
    
    if (b >= batch_size || oc >= out_channels || h >= output_height || w >= output_width) 
        return;
    
    float sum = biases[oc];

    for (int ic = 0; ic < in_channels; ic++) {
        for (int kh = 0; kh < kernel_size; kh++) {
            for (int kw = 0; kw < kernel_size; kw++) {
                int ih = h * stride - padding + kh;
                int iw = w * stride - padding + kw;
                
                if (ih >= 0 && ih < height && iw >= 0 && iw < width) {
                    int input_idx = ((b * in_channels + ic) * height + ih) * width + iw;
                    int weight_idx = ((oc * in_channels + ic) * kernel_size + kh) * kernel_size + kw;
                    sum += input[input_idx] * weights[weight_idx];

                }
            }
        }
    }
    
    int output_idx = ((b * out_channels + oc) * output_height + h) * output_width + w;
    conv_output[output_idx] = sum;
    relu_output[output_idx] = fmaxf(0.0f, sum);
}

__global__ void max_pool_forward_kernel(
    const float* input, float* output, int* pool_indices,
    int batch_size, int channels, int height, int width,
    int pool_size, int pool_stride,
    int output_height, int output_width) {
    
    int b = blockIdx.x; // batch index
    int c = blockIdx.y; // channels
    int h = (blockIdx.z * blockDim.x + threadIdx.x) / output_width; // height
    int w = (blockIdx.z * blockDim.x + threadIdx.x) % output_width; // width
    
    if (b >= batch_size || c >= channels || h >= output_height || w >= output_width) 
        return;
    
    float max_val = -INFINITY;
    int max_idx = -1;
    
    for (int ph = 0; ph < pool_size; ph++) {
        for (int pw = 0; pw < pool_size; pw++) {
            int ih = h * pool_stride + ph;
            int iw = w * pool_stride + pw;
            
            if (ih < height && iw < width) {
                int idx = ((b * channels + c) * height + ih) * width + iw;
                float val = input[idx];
                if (val > max_val) {
                    max_val = val;
                    max_idx = idx;
                }
            }
        }
    }
    
    int output_idx = ((b * channels + c) * output_height + h) * output_width + w;
    output[output_idx] = max_val;
    pool_indices[output_idx] = max_idx;
}

__global__ void conv_backward_kernel(
    const float* grad_output, const float* weights,
    float* grad_input, float* grad_weights, float* grad_biases,
    const float* input, const float* relu_output,
    int batch_size, int in_channels, int out_channels,
    int height, int width, int kernel_size, int stride, int padding,
    int output_height, int output_width) {
    
    int b = blockIdx.x;
    int oc = blockIdx.y;
    int h = blockIdx.z / output_width;
    int w = blockIdx.z % output_width;
    
    if (b >= batch_size || oc >= out_channels || h >= output_height || w >= output_width) 
        return;
    
    int output_idx = ((b * out_channels + oc) * output_height + h) * output_width + w;
    float grad = grad_output[output_idx];
    
    #ifdef DEBUG_PRINT
    if (b == 0 && oc == 0 && h == 0 && w == 0) {
        printf("Conv Backward - Initial gradient: %f\n", grad);
    }
    #endif
    
    // ReLU backward
    if (relu_output[output_idx] <= 0) {
        grad = 0;
    }
    
    // Bias gradient
    atomicAdd(&grad_biases[oc], grad);
    
    
    // Weight and input gradients
    for (int ic = 0; ic < in_channels; ic++) {
        for (int kh = 0; kh < kernel_size; kh++) {
            for (int kw = 0; kw < kernel_size; kw++) {
                int ih = h * stride - padding + kh;
                int iw = w * stride - padding + kw;
                
                if (ih >= 0 && ih < height && iw >= 0 && iw < width) {
                    int input_idx = ((b * in_channels + ic) * height + ih) * width + iw;
                    int weight_idx = ((oc * in_channels + ic) * kernel_size + kh) * kernel_size + kw;
                    
                    atomicAdd(&grad_weights[weight_idx], input[input_idx] * grad);
                    atomicAdd(&grad_input[input_idx], weights[weight_idx] * grad);
                }
            }
        }
    }
}

ConvBlock::ConvBlock(int in_channels, int out_channels, int kernel_size, 
                     int stride, int padding, int pool_size, int pool_stride, 
                     float learning_rate)
    : in_channels(in_channels), out_channels(out_channels), kernel_size(kernel_size),
      stride(stride), padding(padding), pool_size(pool_size), 
      pool_stride(pool_stride), learning_rate(learning_rate), weights_optimizer(learning_rate),
      bias_optimizer(learning_rate),
      d_weights(nullptr), d_biases(nullptr), d_cache(nullptr),
      d_conv_output_cache(nullptr), d_relu_output_cache(nullptr),
      d_pool_indices(nullptr), current_batch_size(0) {

    std::cout << "Initializing ConvBlock with:" << std::endl
              << "  in_channels: " << in_channels << std::endl
              << "  out_channels: " << out_channels << std::endl
              << "  kernel_size: " << kernel_size << std::endl
              << "  stride: " << stride << std::endl
              << "  padding: " << padding << std::endl
              << "  pool_size: " << pool_size << std::endl
              << "  pool_stride: " << pool_stride << std::endl
              << "  learning_rate: " << learning_rate << std::endl;

    if (kernel_size <= 0 || stride <= 0 || padding < 0 || pool_size <= 0 || pool_stride <= 0) {
        throw std::invalid_argument("Invalid convolution parameters");
    }
    
    // Initialize weights and biases
    std::vector<float> h_weights(out_channels * in_channels * kernel_size * kernel_size);
    std::vector<float> h_biases(out_channels);
    
    // Xavier initialization
    float std_dev = sqrt(2.0f / (in_channels * kernel_size * kernel_size));
    std::random_device rd;
    std::mt19937 gen(rd());
    std::normal_distribution<float> distribution(0.0f, std_dev);
    
    for (auto& w : h_weights) {
        w = distribution(gen);
    }
    std::fill(h_biases.begin(), h_biases.end(), 0.01f);

    std::cout << "Initialized weights with std_dev: " << std_dev << std::endl;

    weights_optimizer.init(out_channels * in_channels * kernel_size * kernel_size);
    bias_optimizer.init(out_channels);
    
    // Allocate and copy weights and biases to GPU
    CHECK_CUDA_ERROR(hipMalloc(&d_weights, h_weights.size() * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_biases, h_biases.size() * sizeof(float)));
    
    CHECK_CUDA_ERROR(hipMemcpy(d_weights, h_weights.data(), 
                               h_weights.size() * sizeof(float), 
                               hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_biases, h_biases.data(), 
                               h_biases.size() * sizeof(float), 
                               hipMemcpyHostToDevice));
    
    std::cout << "Successfully allocated and copied weights and biases to GPU" << std::endl;
}

ConvBlock::~ConvBlock() {
    std::cout << "Destroying ConvBlock" << std::endl;
    free_memory();
}

void ConvBlock::allocate_memory(int batch_size) {
    free_memory();
    // Calculate output dimensions
    conv_output_height = (input_height + 2 * padding - kernel_size) / stride + 1;
    conv_output_width = (input_width + 2 * padding - kernel_size) / stride + 1;
    pool_output_height = (conv_output_height - pool_size) / pool_stride + 1;
    pool_output_width = (conv_output_width - pool_size) / pool_stride + 1;
    
    std::cout << "Allocating memory for batch_size: " << batch_size << std::endl
              << "Conv output dimensions: " << conv_output_height << "x" << conv_output_width << std::endl
              << "Pool output dimensions: " << pool_output_height << "x" << pool_output_width << std::endl;
    
    size_t conv_size = batch_size * out_channels * conv_output_height * conv_output_width;
    size_t input_size = batch_size * in_channels * input_height * input_width;
    
    // Allocate memory for intermediate results
    CHECK_CUDA_ERROR(hipMalloc(&d_conv_output_cache, conv_size * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_relu_output_cache, conv_size * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_pool_indices, conv_size * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_cache, input_size * sizeof(float)));
    
    std::cout << "Successfully allocated GPU memory" << std::endl;
}

void ConvBlock::free_memory() {
    std::cout << "Freeing ConvBlock GPU memory" << std::endl;
    if (d_weights) hipFree(d_weights);
    if (d_biases) hipFree(d_biases);
    if (d_cache) hipFree(d_cache);
    if (d_conv_output_cache) hipFree(d_conv_output_cache);
    if (d_relu_output_cache) hipFree(d_relu_output_cache);
    if (d_pool_indices) hipFree(d_pool_indices);
}

void ConvBlock::forward(const float* d_input, float* d_output, int batch_size, int height, int width) {
    input_height = height;
    input_width = width;
    current_batch_size = batch_size;
    
    // Allocate memory for this forward pass
    allocate_memory(batch_size);

    // Copy input to cache
    size_t input_size = batch_size * in_channels * height * width * sizeof(float);
    std::cout << "Copying input to cache, size: " << input_size << " elements" << std::endl;

    CHECK_CUDA_ERROR(hipMemcpy(d_cache, d_input, input_size, hipMemcpyDeviceToDevice));

    dim3 gridDim(batch_size, 
                 out_channels, 
                 (conv_output_height * conv_output_width + 255) / 256);
    dim3 blockDim(256);

    std::cout << "Launching conv forward kernel with grid: " 
              << gridDim.x << "x" << gridDim.y << "x" << gridDim.z 
              << " block: " << blockDim.x << std::endl;

    std::cout << "Batch size: " << batch_size << std::endl;
    std::cout << "Out channels: " << out_channels << std::endl;
    std::cout << "Conv output height: " << conv_output_height << std::endl;
    std::cout << "Conv output width: " << conv_output_width << std::endl;
    std::cout << "In channels: " << in_channels << std::endl;
    std::cout << "Height: " << height << std::endl;
    std::cout << "Width: " << width << std::endl;
    std::cout << "Kernel size: " << kernel_size << std::endl;
    std::cout << "Stride: " << stride << std::endl;
    std::cout << "Padding: " << padding << std::endl;
    std::cout << "Pool size: " << pool_size << std::endl;
    std::cout << "Pool stride: " << pool_stride << std::endl;
    std::cout << "Pool output height: " << pool_output_height << std::endl;
    std::cout << "Pool output width: " << pool_output_width << std::endl;

    conv_forward_kernel<<<gridDim, blockDim>>>(
        d_cache,
        d_weights,
        d_biases,
        d_conv_output_cache,
        d_relu_output_cache,
        batch_size,
        in_channels,
        out_channels,
        height,
        width,
        kernel_size,
        stride,
        padding,
        conv_output_height,
        conv_output_width
    );
    CHECK_LAST_CUDA_ERROR();
    
    // Synchronize and check for errors
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        return;
    }

    dim3 gridDimPooling(batch_size, 
                 out_channels, 
                 (pool_output_height * pool_output_width + 255) / 256);
    dim3 blockDimPooling(256);
    
    // Launch Pooling kernel
    std::cout << "Launching max pooling kernel with grid: " 
              << gridDimPooling.x << "x" << gridDimPooling.y << "x" << gridDimPooling.z 
              << " block: " << blockDimPooling.x << std::endl;
    std::cout << "Batch size: " << batch_size << std::endl;
    std::cout << "Out channels: " << out_channels << std::endl;
    std::cout << "Conv output height: " << conv_output_height << std::endl;
    std::cout << "Conv output width: " << conv_output_width << std::endl;
    std::cout << "Pool size: " << pool_size << std::endl;
    std::cout << "Pool stride: " << pool_stride << std::endl;
    std::cout << "Pool output height: " << pool_output_height << std::endl;
    std::cout << "Pool output width: " << pool_output_width << std::endl;
    std::cout << "Input height: " << height << std::endl;
    std::cout << "Input width: " << width << std::endl;


    max_pool_forward_kernel<<<gridDimPooling, blockDimPooling>>>(
        d_relu_output_cache,
        d_output,
        d_pool_indices,
        batch_size,
        out_channels,
        conv_output_height,
        conv_output_width,
        pool_size,
        pool_stride,
        pool_output_height,
        pool_output_width
    );
    CHECK_LAST_CUDA_ERROR();
    
    // Synchronize and check for errors
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        return;
    }
}

void ConvBlock::backward(const float* d_grad_output, float* d_grad_input, int batch_size) {
    if (batch_size != current_batch_size) {
        throw std::invalid_argument("Batch size mismatch between forward and backward passes");
    }
    
    // Calculate sizes
    size_t weight_size = out_channels * in_channels * kernel_size * kernel_size;
    size_t bias_size = out_channels;
    size_t input_size = batch_size * in_channels * input_height * input_width;
    size_t output_size = batch_size * out_channels * conv_output_height * conv_output_width;
    
    std::cout << "Backward pass dimensions:" << std::endl
              << "  input_size: " << input_size << std::endl
              << "  weight_size: " << weight_size << std::endl
              << "  output_size: " << output_size << std::endl;
    
    // Allocate gradients
    float *d_grad_weights, *d_grad_biases;
    CHECK_CUDA_ERROR(hipMalloc(&d_grad_weights, weight_size * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_grad_biases, bias_size * sizeof(float)));
    
    // Zero out gradients
    CHECK_CUDA_ERROR(hipMemset(d_grad_weights, 0, weight_size * sizeof(float)));
    CHECK_CUDA_ERROR(hipMemset(d_grad_biases, 0, bias_size * sizeof(float)));
    CHECK_CUDA_ERROR(hipMemset(d_grad_input, 0, input_size * sizeof(float)));
    
    // Launch kernel
    int total_spatial_elements = conv_output_height * conv_output_width;
    dim3 gridDim(batch_size, out_channels, (total_spatial_elements + 255) / 256);
    dim3 blockDim(256);

    std::cout << "Launching conv backward kernel with grid: " 
              << gridDim.x << "x" << gridDim.y << "x" << gridDim.z 
              << " block: " << blockDim.x << std::endl;
    std::cout << "Batch size: " << batch_size << std::endl;
    std::cout << "Out channels: " << out_channels << std::endl;
    std::cout << "Conv output height: " << conv_output_height << std::endl;
    std::cout << "Conv output width: " << conv_output_width << std::endl;
    std::cout << "In channels: " << in_channels << std::endl;
    std::cout << "Kernel size: " << kernel_size << std::endl;
    std::cout << "Stride: " << stride << std::endl;
    std::cout << "Padding: " << padding << std::endl;
    std::cout << "Pool size: " << pool_size << std::endl;
    std::cout << "Pool stride: " << pool_stride << std::endl;
    std::cout << "Pool output height: " << pool_output_height << std::endl;
    std::cout << "Pool output width: " << pool_output_width << std::endl;

    conv_backward_kernel<<<gridDim, blockDim>>>(
        d_grad_output, d_weights,
        d_grad_input, d_grad_weights, d_grad_biases,
        d_cache, d_relu_output_cache,
        batch_size, in_channels, out_channels,
        input_height, input_width, kernel_size, stride, padding,
        conv_output_height, conv_output_width
    );

    // Synchronize and check for errors
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        return;
    }
    
    // Update weights and biases
    weights_optimizer.update(d_weights, d_grad_weights);
    bias_optimizer.update(d_biases, d_grad_biases);
    
    // Cleanup
    CHECK_CUDA_ERROR(hipFree(d_grad_weights));
    CHECK_CUDA_ERROR(hipFree(d_grad_biases));
}
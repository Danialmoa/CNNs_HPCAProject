#include "hip/hip_runtime.h"
#include "conv_block.cuh"
#include <random>
#include <cmath>


// CUDA kernels
__global__ void conv_forward_kernel(
    const float* input, const float* weights, const float* biases,
    float* conv_output, float* relu_output,
    int batch_size, int in_channels, int out_channels,
    int height, int width, int kernel_size, int stride, int padding,
    int output_height, int output_width) {
    
    int b = blockIdx.x;
    int oc = blockIdx.y;
    int h = blockIdx.z / output_width;
    int w = blockIdx.z % output_width;
    
    if (b >= batch_size || oc >= out_channels || h >= output_height || w >= output_width) 
        return;
    
    float sum = biases[oc];
    
    for (int ic = 0; ic < in_channels; ic++) {
        for (int kh = 0; kh < kernel_size; kh++) {
            for (int kw = 0; kw < kernel_size; kw++) {
                int ih = h * stride - padding + kh;
                int iw = w * stride - padding + kw;
                
                if (ih >= 0 && ih < height && iw >= 0 && iw < width) {
                    int input_idx = ((b * in_channels + ic) * height + ih) * width + iw;
                    int weight_idx = ((oc * in_channels + ic) * kernel_size + kh) * kernel_size + kw;
                    sum += input[input_idx] * weights[weight_idx];
                }
            }
        }
    }
    
    int output_idx = ((b * out_channels + oc) * output_height + h) * output_width + w;
    conv_output[output_idx] = sum;
    relu_output[output_idx] = fmaxf(0.0f, sum);
}

__global__ void max_pool_forward_kernel(
    const float* input, float* output, int* pool_indices,
    int batch_size, int channels, int height, int width,
    int pool_size, int pool_stride,
    int output_height, int output_width) {
    
    int b = blockIdx.x;
    int c = blockIdx.y;
    int h = blockIdx.z / output_width;
    int w = blockIdx.z % output_width;
    
    if (b >= batch_size || c >= channels || h >= output_height || w >= output_width) 
        return;
    
    float max_val = -INFINITY;
    int max_idx = -1;
    
    for (int ph = 0; ph < pool_size; ph++) {
        for (int pw = 0; pw < pool_size; pw++) {
            int ih = h * pool_stride + ph;
            int iw = w * pool_stride + pw;
            
            if (ih < height && iw < width) {
                int idx = ((b * channels + c) * height + ih) * width + iw;
                float val = input[idx];
                if (val > max_val) {
                    max_val = val;
                    max_idx = idx;
                }
            }
        }
    }
    
    int output_idx = ((b * channels + c) * output_height + h) * output_width + w;
    output[output_idx] = max_val;
    pool_indices[output_idx] = max_idx;
}

__global__ void conv_backward_kernel(
    const float* grad_output, const float* weights,
    float* grad_input, float* grad_weights, float* grad_biases,
    const float* input, const float* relu_output,
    int batch_size, int in_channels, int out_channels,
    int height, int width, int kernel_size, int stride, int padding,
    int output_height, int output_width) {
    
    int b = blockIdx.x;
    int oc = blockIdx.y;
    int h = blockIdx.z / output_width;
    int w = blockIdx.z % output_width;
    
    if (b >= batch_size || oc >= out_channels || h >= output_height || w >= output_width) 
        return;
    
    int output_idx = ((b * out_channels + oc) * output_height + h) * output_width + w;
    float grad = grad_output[output_idx];
    
    // ReLU backward
    if (relu_output[output_idx] <= 0) {
        grad = 0;
    }
    
    // Bias gradient
    atomicAdd(&grad_biases[oc], grad);
    
    // Weight and input gradients
    for (int ic = 0; ic < in_channels; ic++) {
        for (int kh = 0; kh < kernel_size; kh++) {
            for (int kw = 0; kw < kernel_size; kw++) {
                int ih = h * stride - padding + kh;
                int iw = w * stride - padding + kw;
                
                if (ih >= 0 && ih < height && iw >= 0 && iw < width) {
                    int input_idx = ((b * in_channels + ic) * height + ih) * width + iw;
                    int weight_idx = ((oc * in_channels + ic) * kernel_size + kh) * kernel_size + kw;
                    
                    atomicAdd(&grad_weights[weight_idx], input[input_idx] * grad);
                    atomicAdd(&grad_input[input_idx], weights[weight_idx] * grad);
                }
            }
        }
    }
}

ConvBlock::ConvBlock(int in_channels, int out_channels, int kernel_size, 
                     int stride, int padding, int pool_size, int pool_stride, 
                     float learning_rate)
    : in_channels(in_channels), out_channels(out_channels), kernel_size(kernel_size),
      stride(stride), padding(padding), pool_size(pool_size), 
      pool_stride(pool_stride), learning_rate(learning_rate), weights_optimizer(learning_rate),
      bias_optimizer(learning_rate)  {
    
    // Initialize weights and biases
    std::vector<float> h_weights(out_channels * in_channels * kernel_size * kernel_size);
    std::vector<float> h_biases(out_channels);
    
    // Xavier initialization
    float std_dev = sqrt(2.0f / (in_channels * kernel_size * kernel_size));
    std::random_device rd;
    std::mt19937 gen(rd());
    std::normal_distribution<float> distribution(0.0f, std_dev);
    
    for (auto& w : h_weights) {
        w = distribution(gen);
    }
    std::fill(h_biases.begin(), h_biases.end(), 0.01f);

    weights_optimizer.init(out_channels * in_channels * kernel_size * kernel_size);
    bias_optimizer.init(out_channels);
    
    // Allocate and copy weights and biases to GPU
    CHECK_CUDA_ERROR(hipMalloc(&d_weights, h_weights.size() * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_biases, h_biases.size() * sizeof(float)));
    
    CHECK_CUDA_ERROR(hipMemcpy(d_weights, h_weights.data(), 
                               h_weights.size() * sizeof(float), 
                               hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_biases, h_biases.data(), 
                               h_biases.size() * sizeof(float), 
                               hipMemcpyHostToDevice));
}

ConvBlock::~ConvBlock() {
    free_memory();
}

void ConvBlock::allocate_memory(int batch_size) {
    // Calculate output dimensions
    conv_output_height = (input_height + 2 * padding - kernel_size) / stride + 1;
    conv_output_width = (input_width + 2 * padding - kernel_size) / stride + 1;
    pool_output_height = (conv_output_height - pool_size) / pool_stride + 1;
    pool_output_width = (conv_output_width - pool_size) / pool_stride + 1;
    
    size_t conv_size = batch_size * out_channels * conv_output_height * conv_output_width;
    
    // Allocate memory for intermediate results
    CHECK_CUDA_ERROR(hipMalloc(&d_conv_output_cache, conv_size * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_relu_output_cache, conv_size * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_pool_indices, conv_size * sizeof(int)));
}

void ConvBlock::free_memory() {
    if (d_weights) hipFree(d_weights);
    if (d_biases) hipFree(d_biases);
    if (d_cache) hipFree(d_cache);
    if (d_conv_output_cache) hipFree(d_conv_output_cache);
    if (d_relu_output_cache) hipFree(d_relu_output_cache);
    if (d_pool_indices) hipFree(d_pool_indices);
}

void ConvBlock::forward(const float* d_input, float* d_output, 
                       int batch_size, int height, int width) {
    input_height = height;
    input_width = width;
    
    // Allocate memory for this forward pass
    allocate_memory(batch_size);
    
    // Launch convolution + ReLU kernel
    dim3 conv_grid(batch_size, out_channels, conv_output_height * conv_output_width);
    conv_forward_kernel<<<conv_grid, 1>>>(
        d_input, d_weights, d_biases,
        d_conv_output_cache, d_relu_output_cache,
        batch_size, in_channels, out_channels,
        height, width, kernel_size, stride, padding,
        conv_output_height, conv_output_width
    );
    CHECK_LAST_CUDA_ERROR();
    
    // Launch max pooling kernel
    dim3 pool_grid(batch_size, out_channels, pool_output_height * pool_output_width);
    max_pool_forward_kernel<<<pool_grid, 1>>>(
        d_relu_output_cache, d_output, d_pool_indices,
        batch_size, out_channels, conv_output_height, conv_output_width,
        pool_size, pool_stride,
        pool_output_height, pool_output_width
    );
    CHECK_LAST_CUDA_ERROR();
}

void ConvBlock::backward(const float* d_grad_output, float* d_grad_input, int batch_size) {
    // Allocate memory for gradients
    float *d_grad_weights, *d_grad_biases;
    CHECK_CUDA_ERROR(hipMalloc(&d_grad_weights, 
        out_channels * in_channels * kernel_size * kernel_size * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_grad_biases, out_channels * sizeof(float)));
    
    // Zero out gradients
    CHECK_CUDA_ERROR(hipMemset(d_grad_weights, 0, 
        out_channels * in_channels * kernel_size * kernel_size * sizeof(float)));
    CHECK_CUDA_ERROR(hipMemset(d_grad_biases, 0, out_channels * sizeof(float)));
    CHECK_CUDA_ERROR(hipMemset(d_grad_input, 0, 
        batch_size * in_channels * input_height * input_width * sizeof(float)));
    
    // Launch backward kernel
    dim3 grid(batch_size, out_channels, conv_output_height * conv_output_width);
    conv_backward_kernel<<<grid, 1>>>(
        d_grad_output, d_weights,
        d_grad_input, d_grad_weights, d_grad_biases,
        d_cache, d_relu_output_cache,
        batch_size, in_channels, out_channels,
        input_height, input_width, kernel_size, stride, padding,
        conv_output_height, conv_output_width
    );
    CHECK_LAST_CUDA_ERROR();
    
    // Update weights and biases
    // Note: In a real implementation, you would use your optimizer here
    // This is a simple SGD update
    const float update_factor = -learning_rate / batch_size;

    weights_optimizer.update(d_weights, d_grad_weights);
    bias_optimizer.update(d_biases, d_grad_biases);
    
    // Free temporary memory
    hipFree(d_grad_weights);
    hipFree(d_grad_biases);
}
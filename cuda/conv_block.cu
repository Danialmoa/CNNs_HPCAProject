#include "hip/hip_runtime.h"
#include "include/conv_block.cuh"
#include <random>
#include <cmath>
#include <stdexcept>
#include <iostream>


// Performs forward convolution, applies ReLU activation
__global__ void conv_forward_kernel(
    const float* input,           // Input tensor [batch_size, in_channels, height, width]
    const float* weights,         // Weight tensor [out_channels, in_channels, kernel_size, kernel_size] 
    const float* biases,          // Bias tensor [out_channels]
    float* conv_output,           // Raw convolution output
    float* relu_output,           // Output after ReLU activation
    int batch_size,
    int in_channels, 
    int out_channels,
    int height,
    int width, 
    int kernel_size,
    int stride,
    int padding,
    int output_height,
    int output_width) {
    
        __shared__ float shared_input[TILE_SIZE + BLOCK_SIZE - 1][TILE_SIZE + BLOCK_SIZE - 1];
    __shared__ float shared_weights[BLOCK_SIZE][BLOCK_SIZE];

    // Calculate output position
    int b = blockIdx.x;                                    // Batch index
    int oc = blockIdx.y;                                   // Output channel
    int h = (blockIdx.z / ((output_width + TILE_SIZE - 1) / TILE_SIZE)) * TILE_SIZE + threadIdx.y;
    int w = (blockIdx.z % ((output_width + TILE_SIZE - 1) / TILE_SIZE)) * TILE_SIZE + threadIdx.x;

    // Initialize accumulator
    float sum = 0.0f;
    if (threadIdx.x < BLOCK_SIZE && threadIdx.y < BLOCK_SIZE) {
        sum = biases[oc];
    }

    // Loop over input channels
    for (int ic = 0; ic < in_channels; ic++) {
        // Load input tile into shared memory
        for (int i = threadIdx.y; i < TILE_SIZE + kernel_size - 1; i += BLOCK_SIZE) {
            for (int j = threadIdx.x; j < TILE_SIZE + kernel_size - 1; j += BLOCK_SIZE) {
                int ih = h * stride - padding + i;
                int iw = w * stride - padding + j;
                
                if (ih >= 0 && ih < height && iw >= 0 && iw < width && b < batch_size) {
                    shared_input[i][j] = input[((b * in_channels + ic) * height + ih) * width + iw];
                } else {
                    shared_input[i][j] = 0.0f;
                }
            }
        }

        // Load weights into shared memory
        if (threadIdx.y < kernel_size && threadIdx.x < kernel_size) {
            shared_weights[threadIdx.y][threadIdx.x] = 
                weights[((oc * in_channels + ic) * kernel_size + threadIdx.y) * kernel_size + threadIdx.x];
        }

        __syncthreads();

        // Compute convolution for this tile
        if (threadIdx.x < BLOCK_SIZE && threadIdx.y < BLOCK_SIZE && 
            h < output_height && w < output_width && b < batch_size && oc < out_channels) {
            
            for (int kh = 0; kh < kernel_size; kh++) {
                for (int kw = 0; kw < kernel_size; kw++) {
                    int ih = threadIdx.y * stride + kh;
                    int iw = threadIdx.x * stride + kw;
                    sum += shared_input[ih][iw] * shared_weights[kh][kw];
                }
            }
        }

        __syncthreads();
    }

    // Write output
    if (h < output_height && w < output_width && b < batch_size && oc < out_channels) {
        int output_idx = ((b * out_channels + oc) * output_height + h) * output_width + w;
        conv_output[output_idx] = sum;
        
        // ReLU activation
        const float alpha = 0.01f;  // Leaky ReLU slope
        relu_output[output_idx] = sum > 0 ? sum : alpha * sum;
    }
}

// Performs max pooling and tracks indices for backprop
__global__ void max_pool_forward_kernel(
    const float* input,           // Input tensor [batch_size, channels, height, width]
    float* output,                // Pooled output
    int* pool_indices,            // Indices of max values for backprop
    int batch_size,
    int channels,
    int height,
    int width,
    int pool_size,
    int pool_stride,
    int output_height,
    int output_width) {
    
    // Calculate output position
    int b = blockIdx.x;                                    // Batch index
    int c = blockIdx.y;                                    // Channel
    int idx = blockIdx.z * blockDim.x + threadIdx.x;
    int h = idx / output_width;                           // Output height position
    int w = idx % output_width;                           // Output width position

    // Bounds checking
    if (b >= batch_size || c >= channels || h >= output_height || w >= output_width) 
        return;
    
    // Track maximum value and its position
    float max_val = -INFINITY;
    int max_idx = -1;
    
    // Compute max over pooling window
    for (int ph = 0; ph < pool_size; ph++) {
        for (int pw = 0; pw < pool_size; pw++) {
            int ih = h * pool_stride + ph;
            int iw = w * pool_stride + pw;
            
            if (ih < height && iw < width) {
                int idx = ((b * channels + c) * height + ih) * width + iw;
                float val = input[idx];
                if (val > max_val) {
                    max_val = val;
                    max_idx = idx;
                }
            }
        }
    }
    
    // Write outputs
    int output_idx = ((b * channels + c) * output_height + h) * output_width + w;
    output[output_idx] = max_val;
    pool_indices[output_idx] = max_idx;

}

// Computes gradients for convolution layer
__global__ void conv_backward_kernel(
    const float* grad_output,     // Gradient from next layer
    const float* weights,         // Layer weights
    float* grad_input,            // Gradient w.r.t input
    float* grad_weights,          // Gradient w.r.t weights  
    float* grad_biases,           // Gradient w.r.t biases
    const float* input,           // Layer input
    const float* relu_output,     // ReLU activation output
    int batch_size,
    int in_channels,
    int out_channels, 
    int height,
    int width,
    int kernel_size,
    int stride,
    int padding,
    int output_height,
    int output_width) {
    
    // Calculate position
    int b = blockIdx.x;
    int oc = blockIdx.y;
    int h = blockIdx.z / output_width;
    int w = blockIdx.z % output_width;
    
    // Bounds checking
    if (b >= batch_size || oc >= out_channels || h >= output_height || w >= output_width) 
        return;
    
    int output_idx = ((b * out_channels + oc) * output_height + h) * output_width + w;
    float grad = grad_output[output_idx];
    
    // ReLU backward pass - zero out gradient where input was negative
    if (relu_output[output_idx] <= 0) {
        grad = 0;
    }

    // Accumulate bias gradients
    atomicAdd(&grad_biases[oc], grad);
    
    // Compute weight and input gradients
    for (int ic = 0; ic < in_channels; ic++) {
        for (int kh = 0; kh < kernel_size; kh++) {
            for (int kw = 0; kw < kernel_size; kw++) {
                int ih = h * stride - padding + kh;
                int iw = w * stride - padding + kw;
                
                if (ih >= 0 && ih < height && iw >= 0 && iw < width) {
                    int input_idx = ((b * in_channels + ic) * height + ih) * width + iw;
                    int weight_idx = ((oc * in_channels + ic) * kernel_size + kh) * kernel_size + kw;

                    // Accumulate gradients atomically since multiple threads write to same locations
                    atomicAdd(&grad_weights[weight_idx], input[input_idx] * grad);
                    atomicAdd(&grad_input[input_idx], weights[weight_idx] * grad);
                }
            }
        }
    }
}

__global__ void max_pool_backward_kernel(
    const float* grad_output,     // Gradient from next layer
    float* grad_input,            // Gradient to previous layer
    const int* pool_indices,      // Saved max indices from forward pass
    int batch_size,
    int channels,
    int input_height,            // Conv output height
    int input_width,             // Conv output width
    int output_height,           // Pooled output height
    int output_width             // Pooled output width
) {
    // Calculate position
    int b = blockIdx.x;                                    // Batch index
    int c = blockIdx.y;                                    // Channel
    int idx = blockIdx.z * blockDim.x + threadIdx.x;
    int h = idx / output_width;                           // Output height position
    int w = idx % output_width;                           // Output width position
    
    if (b >= batch_size || c >= channels || h >= output_height || w >= output_width) 
        return;
        
    int output_idx = ((b * channels + c) * output_height + h) * output_width + w;
    int input_idx = pool_indices[output_idx];
    
    // Propagate gradient to max element's position
    float grad = grad_output[output_idx];
    atomicAdd(&grad_input[input_idx], grad);
}

// Constructor initializes layer parameters and allocates GPU memory
ConvBlock::ConvBlock(int in_channels, int out_channels, int kernel_size, 
                     int stride, int padding, int pool_size, int pool_stride, 
                     float learning_rate)
    : in_channels(in_channels), out_channels(out_channels), kernel_size(kernel_size),
      stride(stride), padding(padding), pool_size(pool_size), 
      pool_stride(pool_stride), learning_rate(learning_rate), weights_optimizer(learning_rate),
      bias_optimizer(learning_rate),
      d_weights(nullptr), d_biases(nullptr), d_cache(nullptr),
      d_conv_output_cache(nullptr), d_relu_output_cache(nullptr),
      d_pool_indices(nullptr), current_batch_size(0) {

    std::cout << "Initializing ConvBlock with:" << std::endl;
    std::cout << "in_channels: " << in_channels << std::endl;
    std::cout << "out_channels: " << out_channels << std::endl;
    std::cout << "kernel_size: " << kernel_size << std::endl;
    std::cout << "stride: " << stride << std::endl;
    std::cout << "padding: " << padding << std::endl;
    std::cout << "pool_size: " << pool_size << std::endl;
    std::cout << "pool_stride: " << pool_stride << std::endl;
    std::cout << "learning_rate: " << learning_rate << std::endl;

    // Validate parameters
    if (kernel_size <= 0 || stride <= 0 || padding < 0 || pool_size <= 0 || pool_stride <= 0) {
        throw std::invalid_argument("Invalid convolution parameters");
    }
    
    // Initialize weights and biases on CPU
    std::vector<float> h_weights(out_channels * in_channels * kernel_size * kernel_size);
    std::vector<float> h_biases(out_channels);
    
    // Xavier/Glorot initialization for weights
    float std_dev = sqrt(2.0f / (in_channels * kernel_size * kernel_size));
    std::random_device rd;
    std::mt19937 gen(rd());
    std::normal_distribution<float> distribution(0.0f, std_dev);
    
    for (auto& w : h_weights) {
        w = distribution(gen);
    }
    std::fill(h_biases.begin(), h_biases.end(), 0.01f);

    // Initialize optimizers
    weights_optimizer.init(out_channels * in_channels * kernel_size * kernel_size);
    bias_optimizer.init(out_channels);
    
    // Allocate and copy weights and biases to GPU
    CHECK_CUDA_ERROR(hipMalloc(&d_weights, h_weights.size() * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_biases, h_biases.size() * sizeof(float)));
    
    CHECK_CUDA_ERROR(hipMemcpy(d_weights, h_weights.data(), 
                               h_weights.size() * sizeof(float), 
                               hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_biases, h_biases.data(), 
                               h_biases.size() * sizeof(float), 
                               hipMemcpyHostToDevice));
}

// Destructor frees GPU memory
ConvBlock::~ConvBlock() {
    std::cout << "Destroying ConvBlock" << std::endl;
    free_memory();
}

// Allocates GPU memory for intermediate results
void ConvBlock::allocate_memory(int batch_size) {
    // Free any existing allocations
    if (d_conv_output_cache) hipFree(d_conv_output_cache);
    if (d_relu_output_cache) hipFree(d_relu_output_cache);
    if (d_pool_indices) hipFree(d_pool_indices);
    if (d_cache) hipFree(d_cache);

    d_cache = nullptr;
    d_conv_output_cache = nullptr;
    d_relu_output_cache = nullptr;
    d_pool_indices = nullptr;

    // Calculate output dimensions
    conv_output_height = (input_height + 2 * padding - kernel_size) / stride + 1;
    conv_output_width = (input_width + 2 * padding - kernel_size) / stride + 1;
    pool_output_height = (conv_output_height - pool_size) / pool_stride + 1;
    pool_output_width = (conv_output_width - pool_size) / pool_stride + 1;

    size_t conv_size = batch_size * out_channels * conv_output_height * conv_output_width;
    size_t input_size = batch_size * in_channels * input_height * input_width;

    // Allocate memory for intermediate results
    CHECK_CUDA_ERROR(hipMalloc(&d_conv_output_cache, conv_size * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_relu_output_cache, conv_size * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_pool_indices, conv_size * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_cache, input_size * sizeof(float)));

    current_batch_size = batch_size;
}

// Frees all GPU memory
void ConvBlock::free_memory() {
    if (d_weights) {
        hipFree(d_weights);
        d_weights = nullptr;
    }
    if (d_biases) {
        hipFree(d_biases);
        d_biases = nullptr;
    }
    if (d_cache) {
        hipFree(d_cache);
        d_cache = nullptr;
    }
    if (d_conv_output_cache) {
        hipFree(d_conv_output_cache);
        d_conv_output_cache = nullptr;
    }
    if (d_relu_output_cache) {
        hipFree(d_relu_output_cache);
        d_relu_output_cache = nullptr;
    }
    if (d_pool_indices) {
        hipFree(d_pool_indices);
        d_pool_indices = nullptr;
    }
}

// Forward pass: convolution -> ReLU -> max pooling
void ConvBlock::forward(const float* d_input, float* d_output, int batch_size, int height, int width) {

    input_height = height;
    input_width = width;
    current_batch_size = batch_size;
    
    // Allocate memory for this forward pass
    allocate_memory(batch_size);

    // Cache input for backward pass
    size_t input_size = batch_size * in_channels * height * width * sizeof(float);
    CHECK_CUDA_ERROR(hipMemcpy(d_cache, d_input, input_size, hipMemcpyDeviceToDevice));

    // Launch convolution kernel
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(batch_size,
              out_channels,
              ((output_height + TILE_SIZE - 1) / TILE_SIZE) * 
              ((output_width + TILE_SIZE - 1) / TILE_SIZE));

    conv_forward_kernel<<<gridDim, blockDim>>>(
        d_cache,
        d_weights,
        d_biases,
        d_conv_output_cache,
        d_relu_output_cache,
        batch_size,
        in_channels,
        out_channels,
        height,
        width,
        kernel_size,
        stride,
        padding,
        conv_output_height,
        conv_output_width
    );
    CHECK_LAST_CUDA_ERROR();
    
    // Synchronize and check for errors
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error in conv forward: %s\n", hipGetErrorString(err));
        return;
    }

    // Launch pooling kernel
    dim3 gridDimPooling(batch_size, 
                 out_channels, 
                 (pool_output_height * pool_output_width + 255) / 256);
    dim3 blockDimPooling(256);

    max_pool_forward_kernel<<<gridDimPooling, blockDimPooling>>>(
        d_relu_output_cache,
        d_output,
        d_pool_indices,
        batch_size,
        out_channels,
        conv_output_height,
        conv_output_width,
        pool_size,
        pool_stride,
        pool_output_height,
        pool_output_width
    );
    CHECK_LAST_CUDA_ERROR();
    
    // Synchronize and check for errors
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error in pooling forward: %s\n", hipGetErrorString(err));
        return;
    }
}

// Backward pass: computes gradients and updates parameters
void ConvBlock::backward(const float* d_grad_output, float* d_grad_input, int batch_size) {

    if (batch_size != current_batch_size) {
        throw std::invalid_argument("Batch size mismatch between forward and backward passes");
    }
    
    // Calculate sizes
    size_t weight_size = out_channels * in_channels * kernel_size * kernel_size;
    size_t bias_size = out_channels;
    size_t input_size = batch_size * in_channels * input_height * input_width;
 
    // Allocate temporary gradient buffers
    float *d_grad_weights, *d_grad_biases;
    CHECK_CUDA_ERROR(hipMalloc(&d_grad_weights, weight_size * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_grad_biases, bias_size * sizeof(float)));
    
    // Zero out gradients
    CHECK_CUDA_ERROR(hipMemset(d_grad_weights, 0, weight_size * sizeof(float)));
    CHECK_CUDA_ERROR(hipMemset(d_grad_biases, 0, bias_size * sizeof(float)));
    CHECK_CUDA_ERROR(hipMemset(d_grad_input, 0, input_size * sizeof(float)));

    float* d_unpooled_grad;
    size_t conv_output_size = batch_size * out_channels * conv_output_height * conv_output_width;
    CHECK_CUDA_ERROR(hipMalloc(&d_unpooled_grad, conv_output_size * sizeof(float)));
    CHECK_CUDA_ERROR(hipMemset(d_unpooled_grad, 0, conv_output_size * sizeof(float)));
    
    dim3 gridDimPool(batch_size, 
                    out_channels, 
                    (pool_output_height * pool_output_width + 255) / 256);
    dim3 blockDimPool(256);
    
    max_pool_backward_kernel<<<gridDimPool, blockDimPool>>>(
        d_grad_output,
        d_unpooled_grad,
        d_pool_indices,
        batch_size,
        out_channels,
        conv_output_height,
        conv_output_width,
        pool_output_height,
        pool_output_width
    );
    CHECK_LAST_CUDA_ERROR();
 
    
    // Launch backward kernel
    int total_spatial_elements = conv_output_height * conv_output_width;
    dim3 gridDim(batch_size, out_channels, (total_spatial_elements + 255) / 256);
    dim3 blockDim(256);

    conv_backward_kernel<<<gridDim, blockDim>>>(
        d_unpooled_grad, d_weights,
        d_grad_input, d_grad_weights, d_grad_biases,
        d_cache, d_relu_output_cache,
        batch_size, in_channels, out_channels,
        input_height, input_width, kernel_size, stride, padding,
        conv_output_height, conv_output_width
    );

    // Synchronize and check for errors
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error in backward pass: %s\n", hipGetErrorString(err));
        return;
    }
    
    // Update parameters using optimizers
    weights_optimizer.update(d_weights, d_grad_weights);
    bias_optimizer.update(d_biases, d_grad_biases);
    
    // Free temporary buffers
    CHECK_CUDA_ERROR(hipFree(d_grad_weights));
    CHECK_CUDA_ERROR(hipFree(d_grad_biases));
    CHECK_CUDA_ERROR(hipFree(d_unpooled_grad));
}